#include "hip/hip_runtime.h"
//
// Created by max on 29/04/16.
//
#define DEBUG(x) std::cout<<x<<std::endl;
#define STATUS(x) std::cout<<x<<"...";
#define ENDSTATUS std::cout<<"DONE!"<<std::endl;

#include <iostream>
#include <complex>
#include <vector>



#include "../../params/Params1D.hpp"
#include "../TimeOperator1D.hpp"
#include "../TimeOperator.hpp"
#include "../CrankNicolson1D.cpp"

#include "../CNKernels.h"

#define BOOST_TEST_MODULE "DynTest"
#include "boost/test/included/unit_test.hpp"



/*
BOOST_AUTO_TEST_CASE(devicefunctions) {
// check the potential function
    hipDoubleComplex c0 = make_hipDoubleComplex(1.0,0);
    BOOST_CHECK_EQUAL(pot(0).x,c0.x);
    BOOST_CHECK_EQUAL(pot(0).y,c0.y);
// Check the outputs of the mult_rhs method
    const double con1 = 2.0;
    const double con2 = 3.0;
    const double con3 = 4.0;
    const double con4 = 0.0;
    const double con5 = 6.0;
    const double con6 = 0.0;
    hipDoubleComplex c1 = make_hipDoubleComplex(2.0, 2.0);
    hipDoubleComplex c2 = make_hipDoubleComplex(3.0, 3.0);
    hipDoubleComplex c3 = make_hipDoubleComplex(4.0, 4.0);
    hipDoubleComplex c4 = make_hipDoubleComplex(con4, con4);
    hipDoubleComplex c5 = make_hipDoubleComplex(con5, con5);
    hipDoubleComplex c6 = make_hipDoubleComplex(con6, con5);
    hipDoubleComplex h1 = make_hipDoubleComplex(1.0 ,0);
    hipDoubleComplex h2 = make_hipDoubleComplex(1.0, 0);
    mult_rhs( &c3, &c2, &c1, &c5, &c4, &c6, h1, h2 ,0);
    BOOST_CHECK_EQUAL(c5.x,3.0);
    BOOST_CHECK_EQUAL(c5.y,5.0);

    // checking the transform RHS
    hipDoubleComplex s1  = make_hipDoubleComplex(1.0 ,0);
    hipDoubleComplex s2 = make_hipDoubleComplex(3.0,0);
    hipDoubleComplex d;
    const double c_rhs = 5.0;
    const double x_rhs = 0;
    hipDoubleComplex t1 = make_hipDoubleComplex( 1.0, 2.0);
    transform_diag( d, s1, s2, c_rhs, x_rhs,t1);
    BOOST_CHECK_EQUAL(d.x,19.0);
    BOOST_CHECK_EQUAL(d.y,-9.0);
}
*/

BOOST_AUTO_TEST_CASE(BinaryDeviceOperators) {
        hipDoubleComplex res = make_hipDoubleComplex(4.5,1.0);
        hipDoubleComplex l1 = make_hipDoubleComplex(1.0,1.0);
        hipDoubleComplex l2 = make_hipDoubleComplex(2.0,1.0);

        res = l1 + l2 + res;

        BOOST_CHECK_EQUAL(res.x, 7.5);
        BOOST_CHECK_EQUAL(res.y, 3.0);

        std::cout<< res.x<<" "<<res.y<<std::endl;

        res = make_hipDoubleComplex(2.0,1.9);
        l1 = make_hipDoubleComplex(4.0,5.0);
        l2 = make_hipDoubleComplex(5.0,5.0);


        res = res*l1*l2;
        BOOST_CHECK_EQUAL(res.x, -95.5);
        BOOST_CHECK_EQUAL(res.y, 80.5);
        std::cout<< res.x<<" "<<res.y<<std::endl;

        res = make_hipDoubleComplex(2.0,1.9);
        l1 = make_hipDoubleComplex(4.0,5.0);
        res = res / l1;
        BOOST_REQUIRE_CLOSE(res.x, 0.426829268292683, 1e-4);
        BOOST_REQUIRE_CLOSE(res.y, -0.05853658536585366, 1e-4);

        std::cout<< res.x<<" "<<res.y<<std::endl;

        res = make_hipDoubleComplex(2.0,1.9);
        l1 = make_hipDoubleComplex(4.0,5.0);

        res = res - l1;
        BOOST_REQUIRE_CLOSE(res.x, -2.0, 1e-4);
        BOOST_REQUIRE_CLOSE(res.y, -3.1, 1e-4);
}






