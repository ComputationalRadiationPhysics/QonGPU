#include "hip/hip_runtime.h"
//
// Created by max on 29/04/16.
//
#define DEBUG(x) std::cout<<x<<std::endl;
#define STATUS(x) std::cout<<x<<"...";
#define ENDSTATUS std::cout<<"DONE!"<<std::endl;

#include <iostream>
#include <complex>
#include <vector>



#include <params/Params1D.hpp>
#include "../TimeOperator1D.hpp"
#include "../TimeOperator.hpp"
#include "../CrankNicholson1D.cpp"

#include <dynamic/CNKernels.h>

#define BOOST_TEST_MODULE "DynTest"
#include "boost/test/included/unit_test.hpp"


BOOST_AUTO_TEST_CASE(constructor) {

        std::complex<double> xma = 0.0;
    std::complex<double> xmi = 0.0;
    std::complex<double> tmi = 0.0;
    std::complex<double> tma = 0.0;
    const  size_t s1 = 3;
    const size_t s2 = 5;
    const size_t s3 = 6;
    const size_t s4 = 2;
    vector<hipDoubleComplex> v(s1);
    Params1D p( xma, xmi, tma, tmi, s1, s2, s3, s4);
    CrankNicholson1D cn(&p, v);
    BOOST_CHECK_EQUAL(cn.getnx(), s1);
    BOOST_CHECK_EQUAL(cn.getnt(), s3);
    BOOST_CHECK_EQUAL(cn.getxmax(),0.0);
    BOOST_CHECK_EQUAL(cn.getxmin(),0.0);
    BOOST_CHECK_EQUAL(cn.gettmax(),0.0);
    BOOST_CHECK_EQUAL(cn.gettmin(),0.0);
    cn.cusparse_init();
    cn.cusparse_destr();


}

BOOST_AUTO_TEST_CASE(devicefunctions) {
// check the potential function
    hipDoubleComplex c0 = make_hipDoubleComplex(1.0,0);
    BOOST_CHECK_EQUAL(pot(0).x,c0.x);
    BOOST_CHECK_EQUAL(pot(0).y,c0.y);
// Check the outputs of the mult_rhs method
    const double con1 = 2.0;
    const double con2 = 3.0;
    const double con3 = 4.0;
    const double con4 = 0.0;
    const double con5 = 6.0;
    const double con6 = 0.0;
    hipDoubleComplex c1 = make_hipDoubleComplex(2.0, 2.0);
    hipDoubleComplex c2 = make_hipDoubleComplex(3.0, 3.0);
    hipDoubleComplex c3 = make_hipDoubleComplex(4.0, 4.0);
    hipDoubleComplex c4 = make_hipDoubleComplex(con4, con4);
    hipDoubleComplex c5 = make_hipDoubleComplex(con5, con5);
    hipDoubleComplex c6 = make_hipDoubleComplex(con6, con5);
    hipDoubleComplex h1 = make_hipDoubleComplex(1.0 ,0);
    hipDoubleComplex h2 = make_hipDoubleComplex(1.0, 0);
    mult_rhs( &c3, &c2, &c1, &c5, &c4, &c6, h1, h2 ,0);
    BOOST_CHECK_EQUAL(c5.x,3.0);
    BOOST_CHECK_EQUAL(c5.y,5.0);

    // checking the transform RHS
    hipDoubleComplex s1  = make_hipDoubleComplex(1.0 ,0);
    hipDoubleComplex s2 = make_hipDoubleComplex(3.0,0);
    hipDoubleComplex d;
    const double c_rhs = 5.0;
    const double x_rhs = 0;
    hipDoubleComplex t1 = make_hipDoubleComplex( 1.0, 2.0);
    transform_diag( &d, &s1, &s2, c_rhs, x_rhs,t1);
    BOOST_CHECK_EQUAL(d.x,19.0);
    BOOST_CHECK_EQUAL(d.y,-9.0);
}


void dummy(){

}






